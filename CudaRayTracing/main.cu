#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include <iostream>
#include <time.h>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"
#include "pdf.h"

// Disable pedantic warnings for this external library.
#ifdef _MSC_VER
// Microsoft Visual C++ Compiler
#pragma warning(push, 0)
#endif

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// Restore warning levels.
#ifdef _MSC_VER
// Microsoft Visual C++ Compiler
#pragma warning(pop)
#endif

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << ": " << hipGetErrorString(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";

        hipDeviceReset();
        system("pause");
        exit(result);
    }
}

/*
{
	hit_record rec;

	// exceeded the ray bounce limit, no more light gathered
	if (depth <= 0) {
		return color(0, 0, 0);
	}

	// return background color if ray hits nothing
	if (!world.hit(r, 0.001f, infinity, rec)) {
		return background;
	}

	ray scattered;
	color attenuation;
	color emitted = rec.material_ptr->emitted(r, rec, rec.u, rec.v, rec.p);
	float pdf_val;

	if (!rec.material_ptr->scatter(r, rec, attenuation, scattered, pdf_val)) {
		return emitted;
	}

	auto p0 = std::make_shared<hittable_pdf>(rec.p, lights);
	auto p1 = std::make_shared<cosine_pdf>(rec.normal);
	mixture_pdf mixed_pdf(p0, p1);

	scattered = ray(rec.p, mixed_pdf.generate(), r.time());
	pdf_val = mixed_pdf.value(scattered.direction());

	return emitted + attenuation * rec.material_ptr->scattering_pdf(r, rec, scattered) * ray_color(scattered, background, world, lights, depth - 1) / pdf_val;
}
*/

GPU color ray_color(const ray& r, hittable** world, hittable** lights, hiprandState* local_rand) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);

    for(int i = 0; i < 10; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            float pdf_val;
            color emitted = rec.material_ptr->emitted(r, rec, rec.u, rec.v, rec.p);

            if(!rec.material_ptr->scatter(cur_ray, rec, attenuation, scattered, pdf_val, local_rand)) {
                return cur_attenuation * emitted;
            }
            else {
                auto p0 = hittable_pdf(rec.p, *lights);
                auto p1 = cosine_pdf(rec.normal);
                mixture_pdf mixed_pdf(&p0, &p1);

                scattered = ray(rec.p, mixed_pdf.generate(local_rand), r.time());
                pdf_val = mixed_pdf.value(scattered.direction());

                cur_attenuation *= attenuation * rec.material_ptr->scattering_pdf(r, rec, scattered) / pdf_val;
                cur_ray = scattered;
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void create_world(hittable** d_list, hittable** d_world, hittable** lights, camera** cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -1), 0.5f, new lambertian(new solid_color(0.8f, 0.3f, 0.3f)));
        *(d_list + 1) = new sphere(vec3(0, -100.5f, -1), 100, new lambertian(new solid_color(0.8f, 0.8f, 0.2f)));
        *(d_list + 2) = new sphere(vec3(2, 2, -1), 0.25f, new diffuse_light(new solid_color(1.0f, 1.0f, 1.0f)));
        *d_world = new hittable_list(d_list, 3);
        *lights = new sphere(vec3(2, 2, -1), 0.25f, new diffuse_light(new solid_color(1.0f, 1.0f, 1.0f)));

        point3 lookfrom(0, 0.25, 5);
        point3 lookat(0, 0.5, 0);
        vec3 vup(0, 1, 0);
        auto dist_to_focus = 10.0;
        auto aperture = 0.1;
        *cam = new camera(lookfrom, lookat, vup, 45, 12.f/8.f, aperture, (lookat - lookfrom).length());
    }
}

__global__ void render_init(int w, int h, hiprandState* rand) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= w || j >= h) {
        return;
    }

    int pixel = j * w + i;
    hiprand_init(42, pixel, 0, &rand[pixel]);
}

__global__ void render(vec3* fb, int w, int h, int samples, camera** cam, hittable** world, hittable** lights, hiprandState* rand) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= w || j >= h) {
        return;
    }

    // pixel color info
    int pixel = j * w + i;
    hiprandState local_rand = rand[pixel];
    vec3 col(0, 0, 0);
    for (int s = 0; s < samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand)) / float(w);
        float v = float(j + hiprand_uniform(&local_rand)) / float(h);
        ray r = (*cam)->get_ray(u, v, &local_rand);
        col += ray_color(r, world, lights, &local_rand);
    }
    fb[pixel] = col / samples;
}

__global__ void free_world(hittable** d_list, hittable** d_world, hittable** lights, camera** cam) {
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
    delete *lights;
    delete cam;
}

int main() {
    const int width = 1200;
    const int height = 800;
    const int channel_num = 3;
    const int num_pixels = width * height * channel_num;
    size_t fb_size = num_pixels * sizeof(vec3);

    // Setup world
    std::cerr << "Setting up world" << std::endl;
    hittable** obj_list;
    checkCudaErrors(hipMalloc((void**)&obj_list, 2 * sizeof(hittable *)));
    hittable** world;
    checkCudaErrors(hipMalloc((void**)&world, 2 * sizeof(hittable *)));
    camera** cam;
    checkCudaErrors(hipMalloc((void **)&cam, sizeof(camera *)));
    hittable** lights;
    checkCudaErrors(hipMalloc((void**)&lights, sizeof(hittable *)));

    create_world<<<1, 1>>>(obj_list, world, lights, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Setup and render
    std::cerr << "Initializing render" << std::endl;
    hiprandState* rand_state;
    checkCudaErrors(hipMalloc((void**)&rand_state, num_pixels * sizeof(hiprandState)));

    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    const int cr_x = 16;
    const int cr_y = 16;
    const int samples_per_pixel = 1000;
    
    clock_t start, stop;
    start = clock();

    dim3 blocks(width/cr_x + 1, height/cr_y + 1);
    dim3 threads(cr_x, cr_y);
    render_init<<<blocks, threads>>>(width, height, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Starting render" << std::endl;
    render<<<blocks, threads>>>(fb, width, height, samples_per_pixel, cam, world, lights, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Finished render\n";
    std::cerr << "Took " << timer_seconds << " seconds" << std::endl;

    // Frame buffer --> image
    auto* pixels = new unsigned char[width * height * channel_num];
    for (int j = 0; j < height; j++) {
        for (int i = 0; i < width; i++) {
            size_t idx = j * width + i;

            float r = fb[idx].x();
            float g = fb[idx].y();
            float b = fb[idx].z();

			r = std::sqrt(r);
			g = std::sqrt(g);
			b = std::sqrt(b);

            pixels[idx * channel_num] = (unsigned char)(255.99f * r);
            pixels[idx * channel_num + 1] = (unsigned char)(255.99f * g);
            pixels[idx * channel_num + 2] = (unsigned char)(255.99f * b);
        }
    }

    stbi_flip_vertically_on_write(true);
	const int err = stbi_write_jpg("test_image.jpg", width, height, channel_num, pixels, 100);
	if (err) {
		std::cerr << "Image saved successfully\n";
	} else {
		std::cerr << "ERROR::Write_JPG: Image failed to save with code " << err << '\n';
	}

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(obj_list, world, lights, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(obj_list));
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(fb));
    delete[] pixels;

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();

    system("pause");
    return 0;
}