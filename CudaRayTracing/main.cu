#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include <iostream>
#include <time.h>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"
#include "pdf.h"

// Disable pedantic warnings for this external library.
#ifdef _MSC_VER
// Microsoft Visual C++ Compiler
#pragma warning(push, 0)
#endif

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// Restore warning levels.
#ifdef _MSC_VER
// Microsoft Visual C++ Compiler
#pragma warning(pop)
#endif

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << ": " << hipGetErrorString(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";

        hipDeviceReset();
        system("pause");
        exit(result);
    }
}

GPU vec3 ray_color(const ray& r, hittable** world, hittable** lights, hiprandState* local_rand) {
    ray cur_ray = r;
    vec3 cur_attenuation(1.f, 1.f, 1.f);
    int max_depth = 50;
    //vec3 prev_factor(1.f, 1.f, 1.f);

    for(int i = 0; i < max_depth; i++) {
        hit_record rec;
        rec.normal = vec3(0,0,0);
        if ((*world)->hit(cur_ray, 0.001f, infinity, rec)) {
            ray scattered;
            vec3 attenuation;
            color emitted = rec.material_ptr->emitted(r, rec, rec.u, rec.v, rec.p);
	        float pdf_val;

            if (!rec.material_ptr->scatter(cur_ray, rec, attenuation, scattered, pdf_val, local_rand)) {
                return cur_attenuation * emitted;
            } else {
                auto p0 = hittable_pdf(rec.p, *lights);
                auto p1 = cosine_pdf(rec.normal);

                mixture_pdf p(&p0, &p1);
                scattered = ray(rec.p, p.generate(local_rand), r.time());
                pdf_val = p.value(scattered.direction());

                cur_attenuation *= attenuation * rec.material_ptr->scattering_pdf(r, rec, scattered) / pdf_val;;
                cur_ray = scattered;
            }
        } else {
            vec3 unit_direction = cu_unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }

    return vec3(0.0,0.0,0.0);
}

__global__ void create_world(hittable** d_list, hittable** d_world, hittable** lights, camera** cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -1), 0.5f, new lambertian(new solid_color(0.8f, 0.3f, 0.3f)));
        *(d_list + 1) = new sphere(vec3(0, -100.5f, -1), 100, new metal(new solid_color(0.8f, 0.8f, 0.2f), 0.1f));
        *(d_list + 2) = new sphere(vec3(0, 2, -1), 0.25f, new diffuse_light(new solid_color(1.0f, 1.0f, 1.0f)));
        *d_world = new hittable_list(d_list, 3);
        *lights = new sphere(vec3(0, 2, -1), 0.25f, new diffuse_light(new solid_color(5.0f, 5.0f, 5.0f)));

        point3 lookfrom(0, 0.25, 5);
        point3 lookat(0, 0.5, 0);
        vec3 vup(0, 1, 0);
        auto dist_to_focus = 10.0;
        auto aperture = 0.1;
        *cam = new camera(lookfrom, lookat, vup, 45, 12.f/8.f, aperture, (lookat - lookfrom).length());
    }
}

__global__ void render_init(int w, int h, hiprandState* rand) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= w || j >= h) {
        return;
    }

    int pixel = j * w + i;
    hiprand_init(42, pixel, 0, &rand[pixel]);
}

__global__ void render(vec3* fb, int w, int h, int samples, camera** cam, hittable** world, hittable** lights, hiprandState* rand) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= w || j >= h) {
        return;
    }

    // pixel color info
    int pixel = j * w + i;
    hiprandState local_rand = rand[pixel];
    vec3 col(0, 0, 0);
    for (int s = 0; s < samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand)) / float(w);
        float v = float(j + hiprand_uniform(&local_rand)) / float(h);
        ray r = (*cam)->get_ray(u, v, &local_rand);
        col += ray_color(r, world, lights, &local_rand);
    }
    fb[pixel] = col / samples;
}

__global__ void free_world(hittable** d_list, hittable** d_world, hittable** lights, camera** cam) {
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
    delete *lights;
    delete cam;
}

int main() {
    const int width = 1200;
    const int height = 800;
    const int channel_num = 3;
    const int num_pixels = width * height * channel_num;
    size_t fb_size = num_pixels * sizeof(vec3);

    // Setup world
    std::cerr << "Setting up world" << std::endl;
    hittable** obj_list;
    checkCudaErrors(hipMalloc((void**)&obj_list, 2 * sizeof(hittable *)));
    hittable** world;
    checkCudaErrors(hipMalloc((void**)&world, 2 * sizeof(hittable *)));
    camera** cam;
    checkCudaErrors(hipMalloc((void **)&cam, sizeof(camera *)));
    hittable** lights;
    checkCudaErrors(hipMalloc((void**)&lights, sizeof(hittable *)));

    create_world<<<1, 1>>>(obj_list, world, lights, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Setup and render
    std::cerr << "Initializing render" << std::endl;
    hiprandState* rand_state;
    checkCudaErrors(hipMalloc((void**)&rand_state, num_pixels * sizeof(hiprandState)));

    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    const int cr_x = 16;
    const int cr_y = 16;
    
    clock_t start, stop;
    start = clock();

    dim3 blocks(width/cr_x + 1, height/cr_y + 1);
    dim3 threads(cr_x, cr_y);
    render_init<<<blocks, threads>>>(width, height, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Starting render" << std::endl;
    render<<<blocks, threads>>>(fb, width, height, 1000, cam, world, lights, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Finished render\n";
    std::cerr << "Took " << timer_seconds << " seconds" << std::endl;

    // Frame buffer --> image
    auto* pixels = new unsigned char[width * height * channel_num];
    for (int j = 0; j < height; j++) {
        for (int i = 0; i < width; i++) {
            size_t idx = j * width + i;
            pixels[idx * channel_num] = (unsigned char)(255.99f * fb[idx].x());
            pixels[idx * channel_num + 1] = (unsigned char)(255.99f * fb[idx].y());
            pixels[idx * channel_num + 2] = (unsigned char)(255.99f * fb[idx].z());
        }
    }

    stbi_flip_vertically_on_write(true);
	const int err = stbi_write_jpg("test_image.jpg", width, height, channel_num, pixels, 100);
	if (err) {
		std::cerr << "Image saved successfully\n";
	} else {
		std::cerr << "ERROR::Write_JPG: Image failed to save with code " << err << '\n';
	}

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(obj_list, world, lights, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(obj_list));
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(fb));
    delete[] pixels;

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();

    system("pause");
    return 0;
}